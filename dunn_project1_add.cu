/*
 * CS-4370-90: Par. Prog. Many-Core GPUs
 * Nathan Dunn
 * Professor Liu
 * 10/4/19
 * Project 1 - Basic Matrix Addition
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

// -------- EDIT THESE --------------
#define N 8 // size of the matrix
#define BLOCK 4 // size of thread block

/**
	Performs vector addition on the GPU.
	dev_a - first matrix to be added
	dev_b - second matrix to be added
	dev_c - result of dev_a + dev_b stored in this matrix
	size - size of input matrices

*/
__global__ void add_matrix_gpu(int *dev_a, int *dev_b, int *dev_c, int size){
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	int index = row * size + column;
	
	if(column < size && row < size){
		dev_c[index] = dev_a[index] + dev_b[index];
	}
}

/**
	Performs vector addition on the CPU.
	a - first matrix to be added
	b - second matrix to be added
	c - result of a + b stored in this matrix
	size - size of input matrices
*/
void add_matrix_cpu(int *a, int *b, int *c, int size){
	
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int index = i * size + j;
			c[index] = a[index] + b[index];
		}
	}
}

/**
	Prints a matrix.
	matrix - matrix to be printed
	size - size of the matrix
*/
void printMatrix(int * matrix, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			printf("%d ", matrix[i * size + j]);
		}
		printf("\n");
	}
	printf("\n");
}

/**
	Verifies that two matrices are equal.
	a - first matrix to be compared
	b - second matrix to be compared
	size - size of the matrix
*/
void verifySum(int *a, int *b, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int index = i * size + j;
			if(a[index] != b[index]){
				goto FAILED;
			}
		}
	}
	
	printf("TEST PASSED!!!\n");
	return;
	
	FAILED: printf("TEST FAILED!!!\n");
}

int main(void){
	
	// define block size and count
	int blockSize = BLOCK;
	int blockCount = ceil(N/double(blockSize)); 
	dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(blockCount, blockCount, 1);
	
	int *a, *b, *c, *d;
	
	int *dev_a, *dev_b, *dev_c;
	
	// allocate memory for matrix A, B, C, D
	a = (int*)malloc(sizeof(int)*N*N);
	b = (int*)malloc(sizeof(int)*N*N);
	c = (int*)malloc(sizeof(int)*N*N);
	d = (int*)malloc(sizeof(int)*N*N);
	
	// initialize arrays a and b
	int init = 1325;
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			int index = i * N + j;
			init = 3125*init%65536;
			a[index] = (init-32768) / 6553;
			b[index] =  init%1000;
		}
	}
	
	// perform CPU matrix addition for gpu addition verification
	add_matrix_cpu(a, b, c, N);
	
	printf("Matrix A:\n");
	printMatrix(a, N);
	printf("\nMatrix B:\n");
	printMatrix(b, N);
	printf("\nCPU Sum of A + B:\n");
	printMatrix(c, N);
	
	printf("Thread Block Count: %d\n", blockCount);
	printf("Starting GPU Computations\n\n");
	
	// allocate device memory
	hipMalloc((void **)(&dev_a), N*N*sizeof(int));
	hipMalloc((void **)(&dev_b), N*N*sizeof(int));
	hipMalloc((void **)(&dev_c), N*N*sizeof(int));
	
	// copy array a,b (system memory) to dev_a, dev_b (device memory)
	hipMemcpy(dev_a,a,N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*N*sizeof(int), hipMemcpyHostToDevice);
	
	// launch kernels
	add_matrix_gpu<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
	
	hipDeviceSynchronize();
	// copy results from GPU back to system memory
	hipMemcpy(d, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	printf("GPU Sum of A + B:\n");
	printMatrix(d, N);
	
	// verify that CPU and GPU addition match
	verifySum(c, d, N);
	
	// free system and device memory
	free(a);
	free(b);
	free(c);
	free(d); 
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}