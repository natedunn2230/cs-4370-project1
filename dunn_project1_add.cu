/*
 * CS-4370-90: Par. Prog. Many-Core GPUs
 * Nathan Dunn
 * Professor Liu
 * 10/4/19
 * Project 1 - Basic Matrix Addition
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 8 // defines the size of matrix (i.e 8x8 16x16 etc...)

/**
	Performs vector addition on the GPU.
	dev_a - first matrix to be added
	dev_b - second matrix to be added
	dev_c - result of dev_a + dev_b stored in this matrix
	size - size of input matrices

*/
__global__ void gpuAdd(int *dev_a, int *dev_b, int *dev_c, int size){
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	
	int index = row * size + column;
	
	if(column < size && row < size){
		dev_c[index] = dev_a[index] + dev_b[index];
	}
}

/**
	Performs vector addition on the CPU.
	a - first matrix to be added
	b - second matrix to be added
	c - result of a + b stored in this matrix
	size - size of input matrices
*/
void cpuAdd(int *a, int *b, int *c, int size){
	
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int index = i * size + j;
			c[index] = a[index] + b[index];
		}
	}
}

/**
	Prints a matrix.
	matrix - matrix to be printed
	size - size of the matrix
*/
void printMatrix(int * matrix, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			printf("%d ", matrix[i * size + j]);
		}
		printf("\n");
	}
	printf("\n");
}

/**
	Verifies that two matrices are equal.
	a - first matrix to be compared
	b - second matrix to be compared
	size - size of the matrix
*/
void verifySum(int *a, int *b, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int index = i * size + j;
			if(a[index] != b[index]){
				goto FAILED;
			}
		}
	}
	
	printf("TEST PASSED!!!\n");
	return;
	
	FAILED: printf("TEST FAILED!!!\n");
}

int main(void){
	
	// define block size and count
	int blockSize = 4;
	int blockCount = ceil(N/double(blockSize)); 
	dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(blockCount, blockCount, 1);
	
	int *a, *b, *c, *d;
	
	int *dev_a, *dev_b, *dev_c;
	
	// allocate memory for matrix A, B
	a = (int*)malloc(sizeof(int)*N*N);
	b = (int*)malloc(sizeof(int)*N*N);
	c = (int*)malloc(sizeof(int)*N*N);
	d = (int*)malloc(sizeof(int)*N*N);
	
	// "fill" array with values
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			int index = i * N + j;
			a[index] = i + j;
			b[index] =  i - j;
		}
	}
	
	// perform CPU matrix addition for gpu addition verification
	cpuAdd(a, b, c, N);
	
	printf("Matrix A:\n");
	printMatrix(a, N);
	printf("\nMatrix B:\n");
	printMatrix(b, N);
	printf("\n CPU Sum of A + B:\n");
	printMatrix(c, N);
	
	// allocate device memory
	hipMalloc((void **)(&dev_a), N*N*sizeof(int));
	hipMalloc((void **)(&dev_b), N*N*sizeof(int));
	hipMalloc((void **)(&dev_c), N*N*sizeof(int));
	
	// copy array a,b (system memory) to dev_a, dev_b (device memory)
	hipMemcpy(dev_a,a,N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*N*sizeof(int), hipMemcpyHostToDevice);
	
	printf("Thread Block Count: %d\n", blockCount);
	printf("Starting GPU Computations\n");
	
	// launch kernels
	gpuAdd<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
	
	hipDeviceSynchronize();
	// copy results from GPU back to system memory
	hipMemcpy(d, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	printf("GPU Sum of A + B:\n");
	printMatrix(d, N);
	
	// verify that CPU and GPU addition match
	verifySum(c, d, N);
	
	// free system and device memory
	free(a);
	free(b);
	free(c);
	free(d); 
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}