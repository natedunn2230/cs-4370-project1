/*
 * CS-4370-90: Par. Prog. Many-Core GPUs
 * Nathan Dunn
 * Professor Liu
 * 10/4/19
 * Project 1 - Basic Matrix Multiplication
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// -------- EDIT THESE --------------
#define N 8 // size of the matrix
#define BLOCK 4 // size of thread block

/**
 * Performs matrix multiplication on the GPU device
 * dev_a - first matrix to be multiplied
 * dev_b - second matrix to be multiplied
 * dev_c - result of a * b is stored in this matrix
 * size - size of the matrix (size * size)
*/
__global__ void MatrixMulKernel(int *dev_a, int *dev_b, int *dev_c, int size){
	
	int row = blockIdx.y*blockDim.y+threadIdx.y;
	int column = blockIdx.x*blockDim.x+threadIdx.x;
	
	if(row < size && column < size){
		int sum = 0;
		
		for(int k = 0; k < size; k++){
			sum += dev_a[row * size + k] * dev_b[k * size + column];
		}
		
		dev_c[row * size + column] = sum;
	}
	
}

/**
  * Performs matrix multiplication on the CPU
  * a - first matrix to be multiplied
  * b - second matrix to be multiplied
  * c - result of a * b is stored in this matrix
  * size - size of the matrix (size * size)
*/
void MatrixMulOnHost(int *a, int *b, int *c, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int sum = 0;
			for(int k = 0; k < size; k++){
				int d = a[i * size + k];
				int e = b[k * size + j];
				
				sum +=  d * e;
			}
			c[i * size + j] = sum;
		}
	}
}


/**
	Prints a matrix.
	matrix - matrix to be printed
	size - size of the matrix
*/
void printMatrix(int * matrix, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			printf("%d ", matrix[i * size + j]);
		}
		printf("\n");
	}
	printf("\n");
}

/**
	Verifies that two matrices are equal.
	a - first matrix to be compared
	b - second matrix to be compared
	size - size of the matrix
*/
void verifyMult(int *a, int *b, int size){
	for(int i = 0; i < size; i++){
		for(int j = 0; j < size; j++){
			int index = i * size + j;
			if(a[index] != b[index]){
				goto FAILED;
			}
		}
	}
	
	printf("TEST PASSED!!!\n");
	return;
	
	FAILED: printf("TEST FAILED!!!\n");
}

int main(void){
	
	// define block size and count
	int blockSize = BLOCK;
	int blockCount = ceil(N/double(blockSize)); 
	dim3 dimBlock(blockSize, blockSize, 1);
    dim3 dimGrid(blockCount, blockCount, 1);
	
	int *a, *b, *c, *d;
	int *dev_a, *dev_b, *dev_c;
	
	// allocate memory for matrix A, B, C, D
	a = (int*)malloc(sizeof(int)*N*N);
	b = (int*)malloc(sizeof(int)*N*N);
	c = (int*)malloc(sizeof(int)*N*N);
	d = (int*)malloc(sizeof(int)*N*N);
	
	// initialize arrays a and b
	int init = 1325;
	for(int i = 0; i < N; i++){
		for(int j = 0; j < N; j++){
			int index = i * N + j;
			init = 3125*init%65536;
			a[index] = (init-32768)/6553;
			b[index] =  init%1000;
		}
	}
	
	// perform CPU matrix multiplication for gpu multiplication verification
	MatrixMulOnHost(a, b, c, N);
	
	printf("Matrix A:\n");
	printMatrix(a, N);
	printf("\nMatrix B:\n");
	printMatrix(b, N);
	printf("\nCPU Multiplication of A * B:\n");
	printMatrix(c, N);
	
	printf("Thread Block Count: %d\n", blockCount);
	printf("Starting GPU Computations\n\n");
	
	
	// allocate device memory
	hipMalloc((void **)(&dev_a), N*N*sizeof(int));
	hipMalloc((void **)(&dev_b), N*N*sizeof(int));
	hipMalloc((void **)(&dev_c), N*N*sizeof(int));
	
	// copy array a,b (system memory) to dev_a, dev_b (device memory)
	hipMemcpy(dev_a,a,N*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*N*sizeof(int), hipMemcpyHostToDevice);
	
	
	// launch kernels
	MatrixMulKernel<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, N);
	
	hipDeviceSynchronize();
	// copy results from GPU back to system memory
	hipMemcpy(d, dev_c, N*N*sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	printf("GPU Multiplication of A * B:\n");
	printMatrix(d, N);
	
	// verify that CPU and GPU multiplication match
	verifyMult(c, d, N);
	
	// free system and device memory
	free(a);
	free(b);
	free(c);
	free(d); 
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	
	return 0;
}